#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <torch/extension.h>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int cuSPARSE_spmm_csr_kernel(int* dA_csrOffsets, 
                             int* dA_columns, 
                             at::Half* dA_values, 
                             at::Half* dB, 
                             float *dC,
                             const long dimM, 
                             const long dimN, 
                             const long nnz)
{
    const long ldb = dimN; 
    const long ldc = dimN; 
    float alpha = 1.0f;
    float beta = 1.0f;

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )

    //以CSR格式创建A矩阵
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, dimM, dimM, nnz,
                    dA_csrOffsets, dA_columns, dA_values,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) )

    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, dimM, dimN, ldb, dB,
                                        HIP_R_16F, HIPSPARSE_ORDER_ROW) )

    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, dimM, dimN, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(handle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                            HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    CHECK_CUDA( hipFree(dBuffer) )
    return EXIT_SUCCESS;
}

torch::Tensor cuSPARSE_spmm_csr(
    torch::Tensor row_offsets,
    torch::Tensor col_indices, 
    torch::Tensor values, 
    torch::Tensor rhs_matrix,
    const long dimM,
    const long dimN,
    const long nnz){

    auto output_matrix = torch::zeros({dimM,dimN}, torch::kCUDA);

    cuSPARSE_spmm_csr_kernel(
        row_offsets.data<int>(),
        col_indices.data<int>(),
        values.data<at::Half>(),
        rhs_matrix.data<at::Half>(),
        output_matrix.data<float>(),
        dimM,
        dimN,
        nnz
    );

    return output_matrix;
}